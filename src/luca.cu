#include "hip/hip_runtime.h"
#include "luca.h"

struct DBColumn {
public:
    int *data;
    int size;

private:
    DBColumn(int *ptr, int dataSize) {
      data = ptr;
      size = dataSize;
    }

public:
    DBColumn(int dataSize) {
      data = new int[dataSize];
      size = dataSize;
    }

    void fill(int value) {
      fill(0, size, value);
    }

    void fill(int start, int end, int value) {
      for (int i=start; i<end; i++)
        data[i] = value;
    }

    void randomize() {
      randomize(0, size, INT_MIN, INT_MAX);
    }

    void randomize(int min, int max) {
      randomize(0, size, min, max);
    }

    void randomize(int start, int end, int min, int max) {
        std::random_device rd;
        std::mt19937 gen(rd()); // Mersenne Twister generator

        // Define a distribution range
        std::uniform_int_distribution<> dis(min, max);

          for (int i=start; i<end; i++)
            data[i] = dis(gen);
        }

    DBColumn *view(int start, int end) {
        assert(start >= 0);
        assert(end > start);
        assert(end <= size);

        return new DBColumn(data + start, end-start);
    }
};

class DBEngine {
  protected:
    DBColumn *col;

  public:
    DBEngine(DBColumn *column): col(column) { }
    virtual int filterCount(int minIncluded, int maxExcluded) = 0;
    virtual double average() = 0;
    virtual int min() = 0;
    virtual int max() = 0;

    virtual const char* name() = 0;

    // Can execute some particular initialization, like allocating data in the GPU
    virtual void init() { }
    // Can copy memory into the GPU
    virtual void copy() { }

    virtual ~DBEngine() {}
};

class DBEngineCPU: public DBEngine {
  public:
    DBEngineCPU(DBColumn *column): DBEngine(column) {  }

    virtual int filterCount(int minIncluded, int maxExcluded) {
        int count = 0;

        for (int i=0; i<col->size; i++) {
          if (col->data[i] >= minIncluded && col->data[i] < maxExcluded)
            count++;
        }

        return count;
    }

    virtual double average() {
        long sum = 0;

        for (int i=0; i<col->size; i++) {
            sum+=col->data[i];
        }

        return sum / (double) col->size;
    }

    virtual int min() {
        int min = INT_MAX;

        for (int i=0; i<col->size; i++) {
          if (col->data[i] < min)
            min = col->data[i];
        }

        return min;
    }
    virtual int max() {
        int max = INT_MIN;

        for (int i=0; i<col->size; i++) {
          if (col->data[i] > max)
            max = col->data[i];
        }

        return max;
    }

    virtual const char* name() {
      return "CPU";
    }
};

class DBEngineCPU4Threads: public DBEngine {
  private:
    DBEngineCPU *cpu1;
    DBEngineCPU *cpu2;
    DBEngineCPU *cpu3;
    DBEngineCPU *cpu4;
    int threadSize;

  public:
    DBEngineCPU4Threads(DBColumn *column): DBEngine(column) {
        assert(column->size % 4 == 0);
        threadSize = column->size / 4;

        cpu1 = new DBEngineCPU(column->view(0, threadSize));
        cpu2 = new DBEngineCPU(column->view(threadSize, threadSize*2));
        cpu3 = new DBEngineCPU(column->view(threadSize*2, threadSize*3));
        cpu4 = new DBEngineCPU(column->view(threadSize*3, threadSize*4));
     }

    virtual int filterCount(int minIncluded, int maxExcluded) {
       std::future<int> fut1 = std::async(&DBEngineCPU::filterCount, cpu1, minIncluded, maxExcluded);
       std::future<int> fut2 = std::async(&DBEngineCPU::filterCount, cpu2, minIncluded, maxExcluded);
       std::future<int> fut3 = std::async(&DBEngineCPU::filterCount, cpu3, minIncluded, maxExcluded);
       std::future<int> fut4 = std::async(&DBEngineCPU::filterCount, cpu4, minIncluded, maxExcluded);

       return (fut1.get() + fut2.get() + fut3.get() + fut4.get());
    }

    virtual double average() {
        std::future<double> fut1 = std::async(&DBEngineCPU::average, cpu1);
        std::future<double> fut2 = std::async(&DBEngineCPU::average, cpu2);
        std::future<double> fut3 = std::async(&DBEngineCPU::average, cpu3);
        std::future<double> fut4 = std::async(&DBEngineCPU::average, cpu4);

        return (fut1.get() + fut2.get() + fut3.get() + fut4.get()) / 4;
    }

    virtual int min() {
        std::future<int> fut1 = std::async(&DBEngineCPU::min, cpu1);
        std::future<int> fut2 = std::async(&DBEngineCPU::min, cpu2);
        std::future<int> fut3 = std::async(&DBEngineCPU::min, cpu3);
        std::future<int> fut4 = std::async(&DBEngineCPU::min, cpu4);

        return std::min(std::min(fut1.get(), fut2.get()), std::min(fut3.get(), fut4.get()));
    }

    virtual int max() {
        std::future<int> fut1 = std::async(&DBEngineCPU::max, cpu1);
        std::future<int> fut2 = std::async(&DBEngineCPU::max, cpu2);
        std::future<int> fut3 = std::async(&DBEngineCPU::max, cpu3);
        std::future<int> fut4 = std::async(&DBEngineCPU::max, cpu4);

        return std::max(std::max(fut1.get(), fut2.get()), std::max(fut3.get(), fut4.get()));
    }

    virtual const char* name() {
      return "CPU 4";
    }
};

class DBEngineNPP: public DBEngine {
  protected:
    int *deviceData;
    bool pinned;
    bool writeCombining;

    static Npp8u* alloc8u(int numElements) {
      Npp8u* pBuffer; // Pointer for the buffer
      hipMalloc(&pBuffer, numElements);

      return pBuffer;
    }

    static Npp32s* alloc32s(int numElements) {
      Npp32s* pBuffer; // Pointer for the buffer
      hipMalloc(&pBuffer, sizeof(Npp32s) * numElements);

      return pBuffer;
    }

  public:
    DBEngineNPP(DBColumn *column, bool pinMemory, bool useWriteCombining): DBEngine(column) { deviceData = NULL; pinned = pinMemory; writeCombining = useWriteCombining; }

    virtual int filterCount(int minIncluded, int maxExcluded) {
        Npp32s bufferSize, countHost;
        nppsCountInRangeGetBufferSize_32s(col->size, &bufferSize);
        Npp8u* pBuffer = alloc8u(bufferSize);
        Npp32s* pCountDevice = alloc32s(1);

        nppsCountInRange_32s(deviceData, col->size, pCountDevice, minIncluded, maxExcluded + 1, pBuffer);
        hipMemcpy(&countHost, pCountDevice, sizeof(Npp32s), hipMemcpyDeviceToHost);

        hipFree(pBuffer);

        return countHost;
    }

    virtual double average() {
        Npp32s bufferSize, meanHost;
        nppsMeanGetBufferSize_32s_Sfs(col->size, &bufferSize);
        Npp8u* pBuffer = alloc8u(bufferSize);
        Npp32s* pMeanDevice = alloc32s(1);

        nppsMean_32s_Sfs(deviceData, col->size, pMeanDevice, 0, pBuffer);
        hipMemcpy(&meanHost, pMeanDevice, sizeof(Npp32s), hipMemcpyDeviceToHost);

        hipFree(pBuffer);

        return meanHost;
    }

    virtual int min() {
        Npp32s bufferSize, minHost;
        nppsMinGetBufferSize_32s(col->size, &bufferSize);
        Npp8u* pBuffer = alloc8u(bufferSize);
        Npp32s* pMinDevice = alloc32s(1);

        nppsMin_32s(deviceData, col->size, pMinDevice, pBuffer);
        hipMemcpy(&minHost, pMinDevice, sizeof(Npp32s), hipMemcpyDeviceToHost);

        hipFree(pBuffer);

        return minHost;
    }

    virtual int max() {
        Npp32s bufferSize, maxHost;
        nppsMaxGetBufferSize_32s(col->size, &bufferSize);
        Npp8u* pBuffer = alloc8u(bufferSize);
        Npp32s* pMaxDevice = alloc32s(1);

        nppsMax_32s(deviceData, col->size, pMaxDevice, pBuffer);
        hipMemcpy(&maxHost, pMaxDevice, sizeof(Npp32s), hipMemcpyDeviceToHost);

        hipFree(pBuffer);

        return maxHost;
    }

    virtual void init() {
        assert(deviceData == NULL);
        if (writeCombining)
          assert(pinned);

        else if (pinned)
            hipHostAlloc(&deviceData, sizeof(int) * col->size, writeCombining ? hipHostMallocWriteCombined : 0);
        else
            hipMalloc(&deviceData, sizeof(int) * col->size);
    }

    virtual void copy() {
        hipMemcpy(deviceData, col->data, sizeof(int) * col->size, hipMemcpyHostToDevice);
    }

    virtual const char* name() {
      return writeCombining ? "NPP Write combining" : (pinned ? "NPP - pinned" : "NPP");
    }
};

class DBEngineNPPInit: public DBEngineNPP {
    public:
        DBEngineNPPInit(DBColumn *column, bool pinMemory, bool useWriteCombining) : DBEngineNPP(column, pinMemory, useWriteCombining) { }

        virtual const char* name() {
          return writeCombining ? "NPP INIT Write combining" : (pinned ? "NPP INIT - pinned" : "NPP INIT");
        }
};

void runTest(DBEngine *engine) {
    auto start = std::chrono::high_resolution_clock::now();

    engine->init();
    auto init = std::chrono::high_resolution_clock::now();
    engine->copy();
    auto copy = std::chrono::high_resolution_clock::now();

    int count = engine->filterCount(-10000, 10000);
    double average = engine->average();
    int min = engine->min();
    int max = engine->max();

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    auto durationInit = std::chrono::duration_cast<std::chrono::milliseconds>(init - start);
    auto durationCopy = std::chrono::duration_cast<std::chrono::milliseconds>(copy - init);
    std::cout << engine->name() << " - Time: " << duration.count() << " ms - Init: " << durationInit.count() << " ms" << " - Copy: " << durationCopy.count() << " - Count: " << count << " - Average: " << average << " - Min: " << min << " - Max: " << max << std::endl;
}

int main(int argc, char** argv) {
    printf("Submission of Luca Venturi\n");

    DBColumn col(100000000);
    printf("Randomizing\n");
    col.randomize();

    printf("Creating engines\n");
    DBEngineCPU cpu(&col);
    DBEngineNPP nppPinned(&col, true, false);
    DBEngineNPP npp(&col, false, false);
    DBEngineCPU4Threads cpu4(&col);
    // Used to start cuda libraries
    DBEngineNPPInit nppInit(&col, false, false);

    runTest(&cpu);
    runTest(&cpu4);
    runTest(&nppInit);
    runTest(&npp);
    runTest(&nppPinned);
}